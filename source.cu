
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#define _PYTHAG(a, b) (a * a + b * b)
#define ULL unsigned long long

extern "C"
{

    __device__ bool check_valid(unsigned long long data)
    {
        int check_dim = 8;
        // printf("?>??? data is: %d\n", data);
        if (data < pow(10, check_dim - 2))
        {
            return false;
        }
        bool res = true;
        int all_visited = 0;
        int digit = 0, count = 0;
        if (data <= pow(10, check_dim - 1))
        {
            all_visited++;
            count++;
        }
        while (data > 0 && count < 10)
        {
            count += 1;
            digit = data % 10;
            data = data / 10;
            all_visited += 1 << digit;
        }
        return all_visited == (1 << check_dim) - 1;
    }

    __device__ void store_data(int row, int col, int h, int w, int dim, ULL & visited){
        ULL mask;
        for (int i=0; i<h;i++){
            for (int j=0;j<w;j++){
                if (row + i >= dim || col + j >= dim){
                    break;
                }
                // 将visited第i位置为0
                mask = ~(1ULL << ((row + i) * dim + col + j));
                visited &= mask;
                // if (row==5 and col==1){
                //     printf("row:%d col:%d i:%d j:%d board: %llu\n", row, col, i, j, visited);
                // }
            }
        }
        return;
    }

    __device__ void find_h_w(int index, int rotate, int& h, int& w){
        int temp = 0;
        switch (index)
        {
        case 0:
            h = 1; w = 4;
            break;
        case 1:
            h = 1; w = 5;
            break;
        case 2:
            h = 2; w = 2;
            break;
        case 3:
            h = 2; w = 3;
            break;
        case 4:
            h = 2; w = 4;
            break;
        case 5:
            h = 2; w = 5;
            break;
        case 6:
            h = 3; w = 3;
            break;
        case 7:
            h = 3; w = 4;
            break;
        default:
            // printf("error");
            h = -1; w = -1;
        }
        if (rotate != 0){
            temp = h;
            h = w;
            w = temp;
        }
        return;
    }

    __device__ bool solve(unsigned long long tid, int rotate, unsigned long long visited, bool print=false){
        unsigned long long temp;
        int digit, h, w, rc_index, row, col;
        for (int i = 0; i < 8; i++)
        {
            digit = tid % 10;
            tid = tid / 10;
            h = 0;
            w = 0;
            find_h_w(digit, rotate % 2, h, w);
            if (h < 0 or w < 0)
            {
                break;
            }
            rc_index = 0;
            temp = visited;
            while (temp && ((temp & 1ULL) == 0))
            {
                temp /= 2;
                rc_index++;
            }
            row = rc_index / 8;
            col = rc_index % 8;
            store_data(row, col, h, w, 8, visited);
            if (print){
                printf("digit: %d, rotate: %d, put block(%d, %d) at (%d, %d), the board is %llu\n", digit, rotate%2, h, w, row+1, col+1, visited);
            }
            rotate /= 2;
        };
        return visited == 0;
    }

    __global__ void auto_player()
    {
        // 1,4 1,5, 2,2 2,3 2,4 2,5 3,3 3,4
        unsigned long long tid = (unsigned long long)blockIdx.x * (unsigned long long)blockDim.x + (unsigned long long)threadIdx.x;
        unsigned long long ori_tid = tid;
        int rotate = tid & ((1 << 9) - 1);
        // unsigned long long tid = 37021654;
        // int rotate = 0b00111111;
        // tid = tid << 8;
        if (tid == 9477543487ULL || tid == 1ULL){
            printf("tid: %llu, rotate: %d, blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", tid, rotate, blockIdx.x, blockDim.x, threadIdx.x);
        }
        tid = tid >> 8;
        if (!check_valid(tid)){
            return;
        }
        unsigned long long visited=-1, temp;

        // set init state
        store_data(4, 1, 1, 1, 8, visited);
        store_data(3, 6, 2, 1, 8, visited);
        store_data(5, 5, 1, 3, 8, visited);
        // store_data(0, 0, 1, 1, 8, visited);
        // store_data(0, 1, 1, 2, 8, visited);
        // store_data(1, 0, 1, 3, 8, visited);
        // printf("init board: %llu\n", visited);

        bool res = solve(tid, rotate, visited);

        if (res){
            printf("find a solution with tid: %llu, x rotate: %d, blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", tid, rotate, blockIdx.x, blockDim.x, threadIdx.x);
            solve(tid, rotate, visited, true);
        }
    }

} // (End of 'extern "C"' here)